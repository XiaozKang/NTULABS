
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(int n) {
    printf("Hello from GPU%d[%d]!\n", n, threadIdx.x);
}

int main(void){
    printf("Hello from CPU!\n");
    cuda_hello<<<1,4>>>(1);
    cuda_hello<<<1,6>>>(2);
    hipDeviceSynchronize();
    return 0;
}
