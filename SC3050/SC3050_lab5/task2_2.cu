
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 4

__global__ void vector_add(int *d_a, int *d_b, int *d_c){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_c[i] = d_a[i] + d_b[i];
}

int main(void){
    int a[N] = {22,13,16,5};
    int b[N] = {5,22,17,37};
    int c[N];

    int *d_a, *d_b, *d_c;

    // Allocate Memory on the GPU
    hipMalloc((void**)&d_a, sizeof(int)*N);
    hipMalloc((void**)&d_b, sizeof(int)*N);
    hipMalloc((void**)&d_c, sizeof(int)*N);
    // Copy memory from CPU into GPU memory allocated
    hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice);
    // Running on 1 threadblock of 4 threads
    vector_add<<<1,N>>>(d_a, d_b, d_c);
    // Copy result vector from GPU memory to CPU host memory
    hipMemcpy(c, d_c, sizeof(int)*N, hipMemcpyDeviceToHost);
    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    // Printing the inputs and outputs of the vector addition
    printf("A\t");
    for( int i = 0; i < N; i++) printf("%d\t", a[i]);
    printf("\nB\t");
    for( int i = 0; i < N; i++) printf("%d\t", b[i]);
    printf("\nC\t");
    for( int i = 0; i < N; i++) printf("%d\t", c[i]);
    printf("\n");
    return 0;
}
