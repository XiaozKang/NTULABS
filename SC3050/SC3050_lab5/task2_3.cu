
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 4

__global__ void dot_prod(int *d_a, int *d_b, int *d_c){
    __shared__ int temp[N];;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    temp[i] = d_a[i] * d_b[i];
    __syncthreads(); //Synchronize threads
    if (i==0){
        int sum = 0;
        for (int j = 0; j < N; j++) sum = sum + temp[j];
        *d_c = sum;
    }
}

int main(void){
    int a[N] = {22,13,16,5};
    int b[N] = {5,22,17,37};
    int c;

    int *d_a, *d_b, *d_c;

    // Allocate Memory on the GPU
    hipMalloc((void**)&d_a, sizeof(int)*N);
    hipMalloc((void**)&d_b, sizeof(int)*N);
    hipMalloc((void**)&d_c, sizeof(int));
    // Copy memory from CPU into GPU memory allocated
    hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice);
    // Running on 1 threadblock of 4 threads
    dot_prod<<<1,N>>>(d_a, d_b, d_c);
    // Copy result vector from GPU memory to CPU host memory
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    // Printing the inputs and outputs of the vector addition
    printf("A\t");
    for( int i = 0; i < N; i++) printf("%d\t", a[i]);
    printf("\nB\t");
    for( int i = 0; i < N; i++) printf("%d\t", b[i]);
    printf("\nAnswer = %d\t", c);
    printf("\n");
    return 0;
}
