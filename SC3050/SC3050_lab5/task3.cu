
#include <hip/hip_runtime.h>
#include <stdio.h>
#define days 7
#define items 4

__global__ void compute(int *d_data, float *d_price, float *d_sales, float *d_total){
    __shared__ float temp[items];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.x * blockDim.x;
    temp[threadIdx.x] = d_data[i] * d_price[threadIdx.x];
    __syncthreads();
    if (i == j){
        float sum = 0.0;
        for (int k = 0; k < items; k++) sum += temp[k];
        d_sales[blockIdx.x] = sum;
    }
    __syncthreads();
    if (i == 0){
    	float total = 0.0;
	for (int k = 0; k < days; k++) total += d_sales[k];
	*d_total = total;
    }
}

int main(void){
    int data[days][items] = { {3,5,2,0},{2,4,5,1},{0,3,3,1},{3,5,4,4},
                               {4,5,5,3},{10,13,21,16},{8,11,15,8}   };
    float price[items] = {29.99,14.99,9.99,24.99};
    float sales[days];
    float total;

    int *d_data; 
    float *d_price, *d_sales, *d_total;

    // Allocate Memory on the GPU
    hipMalloc((void**)&d_data,  sizeof(int)*days*items);
    hipMalloc((void**)&d_price, sizeof(float)*items);
    hipMalloc((void**)&d_sales, sizeof(float)*days);
    hipMalloc((void**)&d_total, sizeof(float));
    // Copy memory from CPU into GPU memory allocated
    hipMemcpy(d_data,  data,  sizeof(int)*days*items, hipMemcpyHostToDevice);
    hipMemcpy(d_price, price, sizeof(float)*items,    hipMemcpyHostToDevice);
    // Running on 1 threadblock of 4 threads
    compute<<<days,items>>>(d_data, d_price, d_sales, d_total);
    // Copy result vector from GPU memory to CPU host memory
    hipMemcpy(sales,  d_sales, sizeof(float)*days, hipMemcpyDeviceToHost);
    hipMemcpy(&total, d_total, sizeof(float),	    hipMemcpyDeviceToHost);
    // Free GPU memory
    hipFree(d_data);
    hipFree(d_price);
    hipFree(d_sales);
    hipFree(d_total);
    // Printing the inputs and outputs of the vector addition
    printf("Sales per day :\t\n");
    for( int i = 0; i < days; i++) printf("%.2f\n", sales[i]);
    printf("Total Sales :\t%.2f\n", total);
    return 0;
}
